
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c) {
  *c = *a + *b;
}

int main() {

  // host copies of variables a, b & c
  int a, b, c;

  // device copies of variables a, b & c
  int *d_a, *d_b, *d_c;

  // Allocate space for device copies of a, b, c
  int size = sizeof(int);
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  c = 0;
  a = 3;
  b = 5;

  // Copy input data from host to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<<1,1>>>(d_a, d_b, d_c);

  // Copy result from device back to host
  hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  if(err!=hipSuccess) {
      printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
  }
  
  printf("result is %d\n",c);

  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
